#include "hip/hip_runtime.h"
// CUDA-C includes
#include "test.h"

#include <cstdio>

__global__ void addAry( int * ary1, int * ary2 )
{
    int indx = threadIdx.x;
    ary1[ indx ] += ary2[ indx ];
}


// Main cuda function

void runCudaPart() {

    int ary1[32];
    int ary2[32];
    int res[32];

    for( int i=0 ; i<32 ; i++ )
    {
        ary1[i] = i;
        ary2[i] = 2*i;
        res[i]=0;
    }

    int * d_ary1, *d_ary2;
    hipMalloc((void**)&d_ary1, 32*sizeof(int));
    hipMalloc((void**)&d_ary2, 32*sizeof(int));


    hipMemcpy((void*)d_ary1, (void*)ary1, 32*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy((void*)d_ary2, (void*)ary2, 32*sizeof(int), hipMemcpyHostToDevice);


    addAry<<<1,32>>>(d_ary1,d_ary2);

    hipMemcpy((void*)res, (void*)d_ary1, 32*sizeof(int), hipMemcpyDeviceToHost);
    for( int i=0 ; i<32 ; i++ )
        printf( "result[%d] = %d\n", i, res[i]);


    hipFree(d_ary1);
    hipFree(d_ary2);
}
