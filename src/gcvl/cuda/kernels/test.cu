#include "hip/hip_runtime.h"
// CUDA-C includes
#include "test.h"
#include "../cudautils.h"

#include <cstdio>

__global__ void addAry( int * ary1, int * ary2 )
{
    int indx = threadIdx.x;
    ary1[ indx ] += ary2[ indx ];
}


// Main cuda function

void runCudaPart() {

    int * ary1 = new int[32];
    int * ary2 = new int[32];
    int * res = new int[32];

    for( int i=0 ; i<32 ; i++ )
    {
        ary1[i] = i;
        ary2[i] = 2*i;
        res[i]=0;
    }

	CUDA_Array<int> d_ary1, d_ary2;
	d_ary1.Initialize(32,ary1);

    int * d_ary1, *d_ary2;
    hipMalloc((void**)&d_ary1, 32*sizeof(int));
    hipMalloc((void**)&d_ary2, 32*sizeof(int));


    hipMemcpy((void*)d_ary1, (void*)ary1, 32*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy((void*)d_ary2, (void*)ary2, 32*sizeof(int), hipMemcpyHostToDevice);


    addAry<<<1,32>>>(d_ary1,d_ary2);

    hipMemcpy((void*)res, (void*)d_ary1, 32*sizeof(int), hipMemcpyDeviceToHost);
    for( int i=0 ; i<32 ; i++ )
        printf( "result[%d] = %d\n", i, res[i]);


    hipFree(d_ary1);
    hipFree(d_ary2);
}
